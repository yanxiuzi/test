/*
  compile this file with :
  nvcc test_cuda_managed_memory.cu -o test_cuda_managed_memory
*/

#include <hip/hip_runtime.h>
#include <unistd.h>
#include <iostream>

#define SIZE 1024

__global__ void add(int n, float *x, float *y)
{
  int index = threadIdx.x;
  int stride = blockDim.x;

  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// CUDA内核函数，将两个数组相加
__global__ void vec_add(int n, float *x, float *y)
{
  int index = threadIdx.x;
  int stride = blockDim.x;

  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

int main(void)
{
  const int N = 1000;
  const int sz = N*sizeof(float);
  float *x, *y;

  // 检查设备是否支持managed memory
  int device;
  hipDeviceProp_t props;
  gpuErrchk(hipGetDevice(&device));
  gpuErrchk(hipGetDeviceProperties(&props, device));
  std::cout << "Running test on : " << props.name << std::endl;

  // manually set memory type to pinned memory.
  // props.managedMemory = false;

  if (props.managedMemory)
  {
    // 设备支持managed memory
    std::cout << "Device use managed memory: " << device << std::endl;
    gpuErrchk(hipMallocManaged(&x, sz));
    gpuErrchk(hipMallocManaged(&y, sz));
  }
  else
  {
    // 设备不支持managed memory，使用pinned memory
    std::cout << "Device use pinned memory: " << device << std::endl;
    gpuErrchk(hipHostMalloc((void**)&x, sz, hipHostMallocDefault));
    gpuErrchk(hipHostMalloc((void**)&y, sz, hipHostMallocDefault));
  }

  // 初始化x和y数组
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  if (props.managedMemory)
  {
    /// NOTE: jetson不支持 cudaMemPrefetchAsync预取内存, 所以使用cudaStreamAttachMemAsync
    // Prefetch x,y to GPU as they are needed in computation
    gpuErrchk(hipStreamAttachMemAsync(NULL, x, sz, hipMemAttachGlobal));
    gpuErrchk(hipStreamAttachMemAsync(NULL, y, sz, hipMemAttachGlobal));
  }

  // 执行内核
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;
  vec_add<<<numBlocks, blockSize>>>(N, x, y);
  // 检查内核启动是否有错误
  gpuErrchk( hipPeekAtLastError() );

  if (props.managedMemory)
  {
    // Prefetch 'y' to CPU as only 'y' is needed
    gpuErrchk(hipStreamAttachMemAsync(NULL, y, sz, hipMemAttachHost));
  }

  // 等待GPU完成工作
  gpuErrchk(hipDeviceSynchronize());

  // 检查结果
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // 释放内存
  if (props.managedMemory)
  {
    hipFree(x);
    hipFree(y);
  }
  else
  {
    hipHostFree(x);
    hipHostFree(y);
  }

  return 0;
}
